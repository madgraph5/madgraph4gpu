#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <iostream>

#include "mgOnGpuConfig.h"
#include "mgOnGpuTypes.h"

#include "rambo.h"

// Simplified rambo version for 2 to N (with N>=2) processes with massless particles
#ifdef __HIPCC__
namespace grambo2toNm0
#else
namespace rambo2toNm0
#endif
{

  //--------------------------------------------------------------------------

  // Fill in the momenta of the initial particles
  // [NB: the output buffer includes both initial and final momenta, but only initial momenta are filled in]
  __global__
  void getMomentaInitial( const fptype energy, // input: energy
                          fptype momenta1d[]   // output: momenta as AOSOA[npagM][npar][4][neppM]
#ifndef __HIPCC__
                          , const int nevt     // input: #events (for cuda: nevt == ndim == gpublocks*gputhreads)
#endif
                          )
  {
    const int neppM = mgOnGpu::neppM; // ASA layout: constant at compile-time
    fptype (*momenta)[npar][np4][neppM] = (fptype (*)[npar][np4][neppM]) momenta1d; // cast to multiD array pointer (AOSOA)
    const fptype energy1 = energy/2;
    const fptype energy2 = energy/2;
    const fptype mom = energy/2;
#ifndef __HIPCC__
    // ** START LOOP ON IEVT **
    for (int ievt = 0; ievt < nevt; ++ievt)
#endif
    {
#ifdef __HIPCC__
      const int idim = blockDim.x * blockIdx.x + threadIdx.x; // event# == threadid
      const int ievt = idim;
      //printf( "getMomentaInitial: ievt %d\n", ievt );
#endif
      const int ipagM = ievt/neppM; // #eventpage in this iteration
      const int ieppM = ievt%neppM; // #event in the current eventpage in this iteration
      momenta[ipagM][0][0][ieppM] = energy1;
      momenta[ipagM][0][1][ieppM] = 0;
      momenta[ipagM][0][2][ieppM] = 0;
      momenta[ipagM][0][3][ieppM] = mom;
      momenta[ipagM][1][0][ieppM] = energy2;
      momenta[ipagM][1][1][ieppM] = 0;
      momenta[ipagM][1][2][ieppM] = 0;
      momenta[ipagM][1][3][ieppM] = -mom;
    }
    // ** END LOOP ON IEVT **
  }

  //--------------------------------------------------------------------------

  // Fill in the momenta of the final particles using the RAMBO algorithm
  // [NB: the output buffer includes both initial and final momenta, but only initial momenta are filled in]
  __global__
  void getMomentaFinal( const fptype energy,      // input: energy
                        const fptype rnarray1d[], // input: random numbers in [0,1] as AOSOA[npagR][nparf][4][neppR]
                        fptype momenta1d[],       // output: momenta as AOSOA[npagM][npar][4][neppM]
                        fptype wgts[]             // output: weights[nevt]
#ifndef __HIPCC__
                        , const int nevt          // input: #events (for cuda: nevt == ndim == gpublocks*gputhreads)
#endif
                        )
  {
    /****************************************************************************
     *                       rambo                                              *
     *    ra(ndom)  m(omenta)  b(eautifully)  o(rganized)                       *
     *                                                                          *
     *    a democratic multi-particle phase space generator                     *
     *    authors:  s.d. ellis,  r. kleiss,  w.j. stirling                      *
     *    this is version 1.0 -  written by r. kleiss                           *
     *    -- adjusted by hans kuijf, weights are logarithmic (1990-08-20)       *
     *    -- adjusted by madgraph@sheffield_gpu_hackathon team (2020-07-29)     *
     *                                                                          *
     ****************************************************************************/

    const int neppR = mgOnGpu::neppR; // ASA layout: constant at compile-time
    fptype (*rnarray)[nparf][np4][neppR] = (fptype (*)[nparf][np4][neppR]) rnarray1d; // cast to multiD array pointer (AOSOA)
    const int neppM = mgOnGpu::neppM; // ASA layout: constant at compile-time
    fptype (*momenta)[npar][np4][neppM] = (fptype (*)[npar][np4][neppM]) momenta1d; // cast to multiD array pointer (AOSOA)
    
    // initialization step: factorials for the phase space weight
    const fptype twopi = 8. * atan(1.);
    const fptype po2log = log(twopi / 4.);
    fptype z[nparf];
    z[1] = po2log;
    for (int kpar = 2; kpar < nparf; kpar++)
      z[kpar] = z[kpar - 1] + po2log - 2. * log(fptype(kpar - 1));
    for (int kpar = 2; kpar < nparf; kpar++)
      z[kpar] = (z[kpar] - log(fptype(kpar)));

#ifndef __HIPCC__
    // ** START LOOP ON IEVT **
    for (int ievt = 0; ievt < nevt; ++ievt)
#endif
    {
#ifdef __HIPCC__
      const int idim = blockDim.x * blockIdx.x + threadIdx.x; // event# == threadid
      const int ievt = idim;
      //printf( "getMomentaFinal:   ievt %d\n", ievt );
#endif

      const int ipagR = ievt/neppR; // #eventpage in this iteration
      const int ieppR = ievt%neppR; // #event in the current eventpage in this iteration
      const int ipagM = ievt/neppM; // #eventpage in this iteration
      const int ieppM = ievt%neppM; // #event in the current eventpage in this iteration

      fptype& wt = wgts[ievt];

      // generate n massless momenta in infinite phase space
      fptype q[nparf][np4];
      for (int iparf = 0; iparf < nparf; iparf++) {
        const fptype r1 = rnarray[ipagR][iparf][0][ieppR];
        const fptype r2 = rnarray[ipagR][iparf][1][ieppR];
        const fptype r3 = rnarray[ipagR][iparf][2][ieppR];
        const fptype r4 = rnarray[ipagR][iparf][3][ieppR];
        const fptype c = 2. * r1 - 1.;
        const fptype s = sqrt(1. - c * c);
        const fptype f = twopi * r2;
        q[iparf][0] = -log(r3 * r4);
        q[iparf][3] = q[iparf][0] * c;
        q[iparf][2] = q[iparf][0] * s * cos(f);
        q[iparf][1] = q[iparf][0] * s * sin(f);
      }

      // calculate the parameters of the conformal transformation
      fptype r[np4];
      fptype b[np4-1];
      for (int i4 = 0; i4 < np4; i4++)
        r[i4] = 0.;
      for (int iparf = 0; iparf < nparf; iparf++) {
        for (int i4 = 0; i4 < np4; i4++)
          r[i4] = r[i4] + q[iparf][i4];
      }
      const fptype rmas = sqrt(pow(r[0], 2) - pow(r[3], 2) - pow(r[2], 2) - pow(r[1], 2));
      for (int i4 = 1; i4 < np4; i4++)
        b[i4-1] = -r[i4] / rmas;
      const fptype g = r[0] / rmas;
      const fptype a = 1. / (1. + g);
      const fptype x0 = energy / rmas;

      // transform the q's conformally into the p's (i.e. the 'momenta')
      for (int iparf = 0; iparf < nparf; iparf++) {
        fptype bq = b[0] * q[iparf][1] + b[1] * q[iparf][2] + b[2] * q[iparf][3];
        for (int i4 = 1; i4 < np4; i4++)
          momenta[ipagM][iparf+npari][i4][ieppM] = x0 * (q[iparf][i4] + b[i4-1] * (q[iparf][0] + a * bq));
        momenta[ipagM][iparf+npari][0][ieppM] = x0 * (g * q[iparf][0] + bq);
      }

      // calculate weight (NB return log of weight)
      wt = po2log;
      if (nparf != 2)
        wt = (2. * nparf - 4.) * log(energy) + z[nparf-1];

#ifndef __HIPCC__
      // issue warnings if weight is too small or too large
      static int iwarn[5] = {0,0,0,0,0};
      if (wt < -180.) {
        if (iwarn[0] <= 5)
          std::cout << "Too small wt, risk for underflow: " << wt << std::endl;
        iwarn[0] = iwarn[0] + 1;
      }
      if (wt > 174.) {
        if (iwarn[1] <= 5)
          std::cout << "Too large wt, risk for overflow: " << wt << std::endl;
        iwarn[1] = iwarn[1] + 1;
      }
#endif

      // return for weighted massless momenta
      // nothing else to do in this event if all particles are massless (nm==0)

    }
    // ** END LOOP ON IEVT **

    return;
  }


#if defined MGONGPU_CURAND_ONHOST or defined MGONGPU_CURAND_ONDEVICE
  //--------------------------------------------------------------------------

  // Create and initialise a hiprand generator
  void createGenerator( hiprandGenerator_t* pgen )
  {
    // [NB Timings are for GenRnGen host|device (cpp|cuda) generation of 256*32*1 events with nproc=1: rn(0) is host=0.0012s]
    const hiprandRngType_t type = HIPRAND_RNG_PSEUDO_MTGP32;          // 0.00082s | 0.00064s (FOR FAST TESTS)
    //const hiprandRngType_t type = HIPRAND_RNG_PSEUDO_XORWOW;        // 0.049s   | 0.0016s 
    //const hiprandRngType_t type = HIPRAND_RNG_PSEUDO_MRG32K3A;      // 0.71s    | 0.0012s  (better but slower, especially in c++)
    //const hiprandRngType_t type = HIPRAND_RNG_PSEUDO_MT19937;       // 21s      | 0.021s
    //const hiprandRngType_t type = HIPRAND_RNG_PSEUDO_PHILOX4_32_10; // 0.024s   | 0.00026s (used to segfault?)
#if defined __HIPCC__ and defined MGONGPU_CURAND_ONDEVICE
    checkCurand( hiprandCreateGenerator( pgen, type ) );
#else
    checkCurand( hiprandCreateGeneratorHost( pgen, type ) );
#endif
    //checkCurand( hiprandSetGeneratorOrdering( *pgen, HIPRAND_ORDERING_PSEUDO_LEGACY ) ); // CUDA 11
    checkCurand( hiprandSetGeneratorOrdering( *pgen, HIPRAND_ORDERING_PSEUDO_BEST ) );
  }

  //--------------------------------------------------------------------------

  // Seed a hiprand generator
  void seedGenerator( hiprandGenerator_t gen, unsigned long long seed )
  {
    checkCurand( hiprandSetPseudoRandomGeneratorSeed( gen, seed ) );
  }

  //--------------------------------------------------------------------------

  // Destroy a hiprand generator
  void destroyGenerator( hiprandGenerator_t gen )
  {
    checkCurand( hiprandDestroyGenerator( gen ) );
  }

  //--------------------------------------------------------------------------

  // Bulk-generate (using hiprand) the random numbers needed to process nevt events in rambo
  // ** NB: the random numbers are always produced in the same order and are interpreted as an AOSOA
  // AOSOA: rnarray[npagR][nparf][np4][neppR] where nevt=npagR*neppR
  void generateRnarray( hiprandGenerator_t gen, // input: hiprand generator
                        fptype rnarray1d[],    // input: random numbers in [0,1] as AOSOA[npagR][nparf][4][neppR]
                        const int nevt )       // input: #events
  {
#if defined MGONGPU_FPTYPE_DOUBLE
    checkCurand( hiprandGenerateUniformDouble( gen, rnarray1d, np4*nparf*nevt ) );
#elif defined MGONGPU_FPTYPE_FLOAT
    checkCurand( hiprandGenerateUniform( gen, rnarray1d, np4*nparf*nevt ) );
#endif
  }

  //--------------------------------------------------------------------------
#endif

}

